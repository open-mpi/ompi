#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023 The University of Tennessee and The University
 *                         of Tennessee Research Foundation.  All rights
 *                         reserved.
 * Copyright (c) 2020      Research Organization for Information Science
 *                         and Technology (RIST).  All rights reserved.
 * $COPYRIGHT$
 *
 * Additional copyrights may follow
 *
 * $HEADER$
 */

#include "op_cuda_impl.h"

#include <limits.h>

#include <type_traits>

#define ISSIGNED(x) std::is_signed_v<x>
#define ALIGN(x,a,t)            (((x)+((t)(a)-1)) & ~(((t)(a)-1)))
#define ALIGN_PTR(x,a,t)        ((t)ALIGN((uintptr_t)x, a, uintptr_t))
#define ALIGN_PAD_AMOUNT(x,s)   ((~((uintptr_t)(x))+1) & ((uintptr_t)(s)+(!(uintptr_t)(s))-1))

template<typename T, size_t N>
struct __align__(sizeof(T)*N) Vec {
    T v[N];

    template<typename... S>
    __device__ Vec(S... l)
    : v{std::forward<S>(l)...}
    { }

    __device__
    T& operator[](size_t i) { return v[i]; }
    __device__
    const T& operator[](size_t i) const { return v[i]; }
};

template<typename T>
static inline __device__ constexpr T tmax(T a, T b) {
    return (a > b) ? a : b;
}

template<typename T>
static inline __device__ constexpr T tmin(T a, T b) {
    return (a < b) ? a : b;
}

template<typename T>
static inline __device__ constexpr T tsum(T a, T b) {
    return a+b;
}

template<typename T>
static inline __device__ constexpr T tprod(T a, T b) {
    return a*b;
}

template<typename T>
static inline __device__ constexpr T tband(T a, T b) {
    return a&b;
}

template<typename T>
static inline __device__ constexpr T tbor(T a, T b) {
    return a|b;
}

template<typename T>
static inline __device__ constexpr T tbxor(T a, T b) {
    return a^b;
}

template<typename T>
static inline __device__ constexpr T tland(T a, T b) {
    return a&&b;
}

template<typename T>
static inline __device__ constexpr T tlor(T a, T b) {
    return a||b;
}

template<typename T>
static inline __device__ constexpr T tlxor(T a, T b) {
    return ((!!a) ^ (!!b)) ? 1 : 0;
}

template<typename V, typename Fn, size_t ... Ns>
__device__
static inline V apply(const V& a, const V& b, Fn&& fn, std::index_sequence<Ns...>) {
    /* apply fn to all members of the vector and return a new vector */
    return {fn(a[Ns], b[Ns])...};
}


template<typename T, size_t N>
static inline __device__ Vec<T, N> vmax(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return (a > b) ? a : b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vmin(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return (a < b) ? a : b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vsum(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a + b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vprod(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a * b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vband(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a & b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vbor(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a | b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vbxor(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a ^ b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vland(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a && b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vlor(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return a || b; }, std::make_index_sequence<N>{});
}

template<typename T, size_t N>
static inline __device__ Vec<T, N> vlxor(const Vec<T, N>& a, const Vec<T, N>& b) {
    return apply(a, b, [](const T&a, const T&b) -> T { return ((!!a) ^ (!!b)) ? 1 : 0; }, std::make_index_sequence<N>{});
}


/* TODO: missing support for
 * - short float (conditional on whether short float is available)
 */

#define USE_VECTORS 1

#define FUNC_FUNC_FN(name, type_name, type, fn)                                                     \
    static __global__ void                                                                          \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const type *__restrict__ in,                   \
                                                     type *__restrict__ inout, int n) {             \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            inout[i] = fn(inout[i], in[i]);                                                         \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_2buff_##name##_##type_name##_submit(const type *in,                                \
                                              type *inout,                                          \
                                              int count,                                            \
                                              int threads_per_block,                                \
                                              int max_blocks,                                       \
                                              hipStream_t stream) {                                    \
        int threads = min(count, threads_per_block);                                                \
        int blocks  = min((count + threads-1) / threads, max_blocks);                               \
        int n = count;                                                                              \
        hipStream_t s = stream;                                                                        \
        ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(in, inout, n);  \
    }

#define FUNC_FUNC(name, type_name, type) FUNC_FUNC_FN(name, type_name, type, current_func)

#if defined(USE_VECTORS)
#define VFUNC_FUNC(name, type_name, type, vlen, vfn, fn)                                                    \
    static __global__ void                                                                                  \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel_v(const type *__restrict__ in,                         \
                                                       type *__restrict__ inout, int n) {                   \
        using vtype = Vec<type, vlen>;                                                                      \
        constexpr const size_t alignment = sizeof(type)*vlen;                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                            \
        const int stride = blockDim.x * gridDim.x;                                                          \
        size_t in_pad = ALIGN_PAD_AMOUNT(in, alignment);                                                    \
        const vtype * inv = ALIGN_PTR(in, alignment, const vtype*);                                         \
        vtype * inoutv = ALIGN_PTR(inout, alignment, vtype*);                                               \
        for (int i = index; i < (n/vlen - in_pad/sizeof(type)); i += stride) {                              \
            inoutv[i] = vfn(inoutv[i], inv[i]);                                                             \
        }                                                                                                   \
        if (in_pad > 0) {                                                                                   \
            /* manage front values */                                                                       \
            if (index < ((in_pad/sizeof(type)) - 1)) {                                                      \
                inout[index] = fn(inout[index], in[index]);                                                 \
            }                                                                                               \
        }                                                                                                   \
        int remainder = (n%vlen);                                                                           \
        if (remainder > 0) {                                                                                \
            /* manage back values */                                                                        \
            if (index < (remainder-1)) {                                                                    \
                size_t idx = n - remainder + index;                                                         \
                inout[idx] = fn(inout[idx], in[idx]);                                                       \
            }                                                                                               \
        }                                                                                                   \
    }                                                                                                       \
    static __global__ void                                                                                  \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const type *__restrict__ in,                           \
                                                     type *__restrict__ inout, int n) {                     \
        /* non-vectorized version (e.g., due to mismatching alignment) */                                   \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                            \
        const int stride = blockDim.x * gridDim.x;                                                          \
        for (int i = index; i < n; i += stride) {                                                           \
            inout[i] = fn(inout[i], in[i]);                                                                 \
        }                                                                                                   \
    }                                                                                                       \
    void                                                                                                    \
    ompi_op_cuda_2buff_##name##_##type_name##_submit(const type *in,                                        \
                                              type *inout,                                                  \
                                              int count,                                                    \
                                              int threads_per_block,                                        \
                                              int max_blocks,                                               \
                                              hipStream_t stream) {                                            \
        int vcount  = (count + vlen-1)/vlen;                                                                \
        int threads = min(threads_per_block, vcount);                                                       \
        int blocks  = min((vcount + threads-1) / threads, max_blocks);                                      \
        int n = count;                                                                                      \
        hipStream_t s = stream;                                                                                \
        constexpr const size_t alignment = sizeof(type)*vlen;                                               \
        size_t in_pad = ALIGN_PAD_AMOUNT(in, alignment);                                                    \
        size_t inout_pad = ALIGN_PAD_AMOUNT(inout, alignment);                                              \
        if (in_pad == inout_pad) {                                                                          \
            ompi_op_cuda_2buff_##name##_##type_name##_kernel_v<<<blocks, threads, 0, s>>>(in, inout, n);    \
        } else {                                                                                            \
            ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(in, inout, n);      \
        }   \
    }
#else
#define VFUNC_FUNC(name, type_name, type, vlen, vfn, fn) FUNC_FUNC_FN(name, type_name, type, fn)
#endif // defined(USE_VECTORS)

/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for minloc and maxloc
 */

#define LOC_FUNC(name, type_name, op)                                                               \
    static __global__ void                                                                          \
    ompi_op_cuda_2buff_##name##_##type_name##_kernel(const ompi_op_predefined_##type_name##_t *__restrict__ in,  \
                                                     ompi_op_predefined_##type_name##_t *__restrict__ inout,     \
                                                     int n)                                         \
    {                                                                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            const ompi_op_predefined_##type_name##_t *a = &in[i];                                   \
            ompi_op_predefined_##type_name##_t *b = &inout[i];                                      \
            if (a->v op b->v) {                                                                     \
                b->v = a->v;                                                                        \
                b->k = a->k;                                                                        \
            } else if (a->v == b->v) {                                                              \
                b->k = (b->k < a->k ? b->k : a->k);                                                 \
            }                                                                                       \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_2buff_##name##_##type_name##_submit(const ompi_op_predefined_##type_name##_t *a,   \
                                            ompi_op_predefined_##type_name##_t *b,                  \
                                            int count,                                              \
                                            int threads_per_block,                                  \
                                            int max_blocks,                                         \
                                            hipStream_t stream) {                                      \
        int threads = min(count, threads_per_block);                                                \
        int blocks  = min((count + threads-1) / threads, max_blocks);                               \
        hipStream_t s = stream;                                                                        \
        ompi_op_cuda_2buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(a, b, count);   \
    }

/*************************************************************************
 * Max
 *************************************************************************/

/* C integer */

/* fixed-size types: 16B vector sizes
 * TODO: should this be fine-tuned to the architecture? */
VFUNC_FUNC(max,   int8_t,   int8_t, 16, vmax, tmax)
VFUNC_FUNC(max,  uint8_t,  uint8_t, 16, vmax, tmax)
VFUNC_FUNC(max,  int16_t,  int16_t, 8, vmax, tmax)
VFUNC_FUNC(max, uint16_t, uint16_t, 8, vmax, tmax)
VFUNC_FUNC(max,  int32_t,  int32_t, 4, vmax, tmax)
VFUNC_FUNC(max, uint32_t, uint32_t, 4, vmax, tmax)
VFUNC_FUNC(max,  int64_t,  int64_t, 2, vmax, tmax)
VFUNC_FUNC(max, uint64_t, uint64_t, 2, vmax, tmax)

VFUNC_FUNC(max,  long,  long, 2, vmax, tmax)
VFUNC_FUNC(max, ulong, unsigned long, 2, vmax, tmax)

/* float */
VFUNC_FUNC(max, float, float, 4, vmax, tmax)
VFUNC_FUNC(max, double, double, 2, vmax, tmax)
VFUNC_FUNC(max, long_double, long double, 1, vmax, tmax)

/*************************************************************************
 * Min
 *************************************************************************/

/* C integer */
VFUNC_FUNC(min,   int8_t,   int8_t, 16, vmin, tmin)
VFUNC_FUNC(min,  uint8_t,  uint8_t, 16, vmin, tmin)
VFUNC_FUNC(min,  int16_t,  int16_t, 8, vmin, tmin)
VFUNC_FUNC(min, uint16_t, uint16_t, 8, vmin, tmin)
VFUNC_FUNC(min,  int32_t,  int32_t, 4, vmin, tmin)
VFUNC_FUNC(min, uint32_t, uint32_t, 4, vmin, tmin)
VFUNC_FUNC(min,  int64_t,  int64_t, 2, vmin, tmin)
VFUNC_FUNC(min, uint64_t, uint64_t, 2, vmin, tmin)
VFUNC_FUNC(min,  long,  long, 2, vmin, tmin)
VFUNC_FUNC(min, ulong, unsigned long, 2, vmin, tmin)

/* float */
VFUNC_FUNC(min, float, float, 4, vmin, tmin)
VFUNC_FUNC(min, double, double, 2, vmin, tmin)
VFUNC_FUNC(min, long_double, long double, 1, vmin, tmin)

/*************************************************************************
 * Sum
 *************************************************************************/

/* C integer */
VFUNC_FUNC(sum,   int8_t,   int8_t, 16, vsum, tsum)
VFUNC_FUNC(sum,  uint8_t,  uint8_t, 16, vsum, tsum)
VFUNC_FUNC(sum,  int16_t,  int16_t, 8, vsum, tsum)
VFUNC_FUNC(sum, uint16_t, uint16_t, 8, vsum, tsum)
VFUNC_FUNC(sum,  int32_t,  int32_t, 4, vsum, tsum)
VFUNC_FUNC(sum, uint32_t, uint32_t, 4, vsum, tsum)
VFUNC_FUNC(sum,  int64_t,  int64_t, 2, vsum, tsum)
VFUNC_FUNC(sum, uint64_t, uint64_t, 2, vsum, tsum)
VFUNC_FUNC(sum,  long,  long, 2, vsum, tsum)
VFUNC_FUNC(sum, ulong, unsigned long, 2, vsum, tsum)

/* float */
VFUNC_FUNC(sum, float, float, 4, vsum, tsum)
VFUNC_FUNC(sum, double, double, 2, vsum, tsum)
VFUNC_FUNC(sum, long_double, long double, 1, vsum, tsum)

/* Complex */
#undef current_func
#define current_func(a, b) (hipCaddf(a,b))
FUNC_FUNC(sum, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCadd(a,b))
FUNC_FUNC(sum, c_double_complex, hipDoubleComplex)

/*************************************************************************
 * Product
 *************************************************************************/

/* C integer */
VFUNC_FUNC(prod,   int8_t,   int8_t, 16, vprod, tprod)
VFUNC_FUNC(prod,  uint8_t,  uint8_t, 16, vprod, tprod)
VFUNC_FUNC(prod,  int16_t,  int16_t, 8, vprod, tprod)
VFUNC_FUNC(prod, uint16_t, uint16_t, 8, vprod, tprod)
VFUNC_FUNC(prod,  int32_t,  int32_t, 4, vprod, tprod)
VFUNC_FUNC(prod, uint32_t, uint32_t, 4, vprod, tprod)
VFUNC_FUNC(prod,  int64_t,  int64_t, 2, vprod, tprod)
VFUNC_FUNC(prod, uint64_t, uint64_t, 2, vprod, tprod)
VFUNC_FUNC(prod,  long,  long, 2, vprod, tprod)
VFUNC_FUNC(prod, ulong, unsigned long, 2, vprod, tprod)

/* float */
VFUNC_FUNC(prod, float, float, 4, vprod, tprod)
VFUNC_FUNC(prod, double, double, 2, vprod, tprod)
VFUNC_FUNC(prod, long_double, long double, 1, vprod, tprod)

/* Complex */
#undef current_func
#define current_func(a, b) (hipCmulf(a,b))
FUNC_FUNC(prod, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCmul(a,b))
FUNC_FUNC(prod, c_double_complex, hipDoubleComplex)

/*************************************************************************
 * Logical AND
 *************************************************************************/

/* C integer */
VFUNC_FUNC(land,   int8_t,   int8_t, 16, vland, tland)
VFUNC_FUNC(land,  uint8_t,  uint8_t, 16, vland, tland)
VFUNC_FUNC(land,  int16_t,  int16_t, 8, vland, tland)
VFUNC_FUNC(land, uint16_t, uint16_t, 8, vland, tland)
VFUNC_FUNC(land,  int32_t,  int32_t, 4, vland, tland)
VFUNC_FUNC(land, uint32_t, uint32_t, 4, vland, tland)
VFUNC_FUNC(land,  int64_t,  int64_t, 2, vland, tland)
VFUNC_FUNC(land, uint64_t, uint64_t, 2, vland, tland)
VFUNC_FUNC(land,  long,  long, 2, vland, tland)
VFUNC_FUNC(land, ulong, unsigned long, 2, vland, tland)

/* C++ bool */
VFUNC_FUNC(land, bool, bool, 16, vland, tland)

/*************************************************************************
 * Logical OR
 *************************************************************************/

/* C integer */
VFUNC_FUNC(lor,   int8_t,   int8_t, 16, vlor, tlor)
VFUNC_FUNC(lor,  uint8_t,  uint8_t, 16, vlor, tlor)
VFUNC_FUNC(lor,  int16_t,  int16_t, 8, vlor, tlor)
VFUNC_FUNC(lor, uint16_t, uint16_t, 8, vlor, tlor)
VFUNC_FUNC(lor,  int32_t,  int32_t, 4, vlor, tlor)
VFUNC_FUNC(lor, uint32_t, uint32_t, 4, vlor, tlor)
VFUNC_FUNC(lor,  int64_t,  int64_t, 2, vlor, tlor)
VFUNC_FUNC(lor, uint64_t, uint64_t, 2, vlor, tlor)
VFUNC_FUNC(lor,  long,  long, 2, vlor, tlor)
VFUNC_FUNC(lor, ulong, unsigned long, 2, vlor, tlor)

/* C++ bool */
VFUNC_FUNC(lor, bool, bool, 16, vlor, tlor)

/*************************************************************************
 * Logical XOR
 *************************************************************************/

/* C integer */
VFUNC_FUNC(lxor,   int8_t,   int8_t, 16, vlxor, tlxor)
VFUNC_FUNC(lxor,  uint8_t,  uint8_t, 16, vlxor, tlxor)
VFUNC_FUNC(lxor,  int16_t,  int16_t, 8, vlxor, tlxor)
VFUNC_FUNC(lxor, uint16_t, uint16_t, 8, vlxor, tlxor)
VFUNC_FUNC(lxor,  int32_t,  int32_t, 4, vlxor, tlxor)
VFUNC_FUNC(lxor, uint32_t, uint32_t, 4, vlxor, tlxor)
VFUNC_FUNC(lxor,  int64_t,  int64_t, 2, vlxor, tlxor)
VFUNC_FUNC(lxor, uint64_t, uint64_t, 2, vlxor, tlxor)
VFUNC_FUNC(lxor,  long,  long, 2, vlxor, tlxor)
VFUNC_FUNC(lxor, ulong, unsigned long, 2, vlxor, tlxor)

/* C++ bool */
VFUNC_FUNC(lxor, bool, bool, 16, vlxor, tlxor)


/*************************************************************************
 * Bitwise AND
 *************************************************************************/

/* C integer */
VFUNC_FUNC(band,   int8_t,   int8_t, 16, vband, tband)
VFUNC_FUNC(band,  uint8_t,  uint8_t, 16, vband, tband)
VFUNC_FUNC(band,  int16_t,  int16_t, 8, vband, tband)
VFUNC_FUNC(band, uint16_t, uint16_t, 8, vband, tband)
VFUNC_FUNC(band,  int32_t,  int32_t, 4, vband, tband)
VFUNC_FUNC(band, uint32_t, uint32_t, 4, vband, tband)
VFUNC_FUNC(band,  int64_t,  int64_t, 2, vband, tband)
VFUNC_FUNC(band, uint64_t, uint64_t, 2, vband, tband)
VFUNC_FUNC(band,  long,  long, 2, vband, tband)
VFUNC_FUNC(band, ulong, unsigned long, 2, vband, tband)

/* C++ byte */
VFUNC_FUNC(band, byte, char, 16, vband, tband)

/*************************************************************************
 * Bitwise OR
 *************************************************************************/

/* C integer */
VFUNC_FUNC(bor,   int8_t,   int8_t, 16, vbor, tbor)
VFUNC_FUNC(bor,  uint8_t,  uint8_t, 16, vbor, tbor)
VFUNC_FUNC(bor,  int16_t,  int16_t, 8, vbor, tbor)
VFUNC_FUNC(bor, uint16_t, uint16_t, 8, vbor, tbor)
VFUNC_FUNC(bor,  int32_t,  int32_t, 4, vbor, tbor)
VFUNC_FUNC(bor, uint32_t, uint32_t, 4, vbor, tbor)
VFUNC_FUNC(bor,  int64_t,  int64_t, 2, vbor, tbor)
VFUNC_FUNC(bor, uint64_t, uint64_t, 2, vbor, tbor)
VFUNC_FUNC(bor,  long,  long, 2, vbor, tbor)
VFUNC_FUNC(bor, ulong, unsigned long, 2, vbor, tbor)

/* C++ byte */
VFUNC_FUNC(bor, byte, char, 16, vbor, tbor)

/*************************************************************************
 * Bitwise XOR
 *************************************************************************/

/* C integer */
VFUNC_FUNC(bxor,   int8_t,   int8_t, 16, vbxor, tbxor)
VFUNC_FUNC(bxor,  uint8_t,  uint8_t, 16, vbxor, tbxor)
VFUNC_FUNC(bxor,  int16_t,  int16_t, 8, vbxor, tbxor)
VFUNC_FUNC(bxor, uint16_t, uint16_t, 8, vbxor, tbxor)
VFUNC_FUNC(bxor,  int32_t,  int32_t, 4, vbxor, tbxor)
VFUNC_FUNC(bxor, uint32_t, uint32_t, 4, vbxor, tbxor)
VFUNC_FUNC(bxor,  int64_t,  int64_t, 2, vbxor, tbxor)
VFUNC_FUNC(bxor, uint64_t, uint64_t, 2, vbxor, tbxor)
VFUNC_FUNC(bxor,  long,  long, 2, vbxor, tbxor)
VFUNC_FUNC(bxor, ulong, unsigned long, 2, vbxor, tbxor)

/* C++ byte */
VFUNC_FUNC(bxor, byte, char, 16, vbxor, tbxor)

/*************************************************************************
 * Max location
 *************************************************************************/

LOC_FUNC(maxloc, float_int, >)
LOC_FUNC(maxloc, double_int, >)
LOC_FUNC(maxloc, long_int, >)
LOC_FUNC(maxloc, 2int, >)
LOC_FUNC(maxloc, short_int, >)
LOC_FUNC(maxloc, long_double_int, >)

/* Fortran compat types */
LOC_FUNC(maxloc, 2float, >)
LOC_FUNC(maxloc, 2double, >)
LOC_FUNC(maxloc, 2int8, >)
LOC_FUNC(maxloc, 2int16, >)
LOC_FUNC(maxloc, 2int32, >)
LOC_FUNC(maxloc, 2int64, >)

/*************************************************************************
 * Min location
 *************************************************************************/

LOC_FUNC(minloc, float_int, <)
LOC_FUNC(minloc, double_int, <)
LOC_FUNC(minloc, long_int, <)
LOC_FUNC(minloc, 2int, <)
LOC_FUNC(minloc, short_int, <)
LOC_FUNC(minloc, long_double_int, <)

/* Fortran compat types */
LOC_FUNC(minloc, 2float, <)
LOC_FUNC(minloc, 2double, <)
LOC_FUNC(minloc, 2int8, <)
LOC_FUNC(minloc, 2int16, <)
LOC_FUNC(minloc, 2int32, <)
LOC_FUNC(minloc, 2int64, <)

/*
 *  This is a three buffer (2 input and 1 output) version of the reduction
 *    routines, needed for some optimizations.
 */
#define OP_FUNC_3BUF(name, type_name, type, op)                                                     \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const type *__restrict__ in1,                  \
                                                     const type *__restrict__ in2,                  \
                                                     type *__restrict__ out, int n) {               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            out[i] = in1[i] op in2[i];                                                              \
        }                                                                                           \
    }                                                                                               \
    void ompi_op_cuda_3buff_##name##_##type_name##_submit(const type *in1, const type *in2,         \
                                                          type *out, int count,                     \
                                                          int threads_per_block,                    \
                                                          int max_blocks,                           \
                                                          hipStream_t stream) {                        \
        int threads = min(count, threads_per_block);                                                \
        int blocks  = min((count + threads-1) / threads, max_blocks);                               \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }


/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for (out = op(in1, in2))
 */
#define FUNC_FUNC_3BUF(name, type_name, type)                                                       \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const type *__restrict__ in1,                  \
                                                     const type *__restrict__ in2,                  \
                                                     type *__restrict__ out, int n) {               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            out[i] = current_func(in1[i], in2[i]);                                                  \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_3buff_##name##_##type_name##_submit(const type *in1, const type *in2,              \
                                                     type *out, int count,                          \
                                                     int threads_per_block,                         \
                                                     int max_blocks,                                \
                                                     hipStream_t stream) {                             \
        int threads = min(count, threads_per_block);                                                \
        int blocks  = min((count + threads-1) / threads, max_blocks);                               \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }


#if defined(USE_VECTORS)
#define VFUNC_FUNC_3BUF(name, type_name, type, vlen, vfn, fn)                                               \
    static __global__ void                                                                                  \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel_v(const type *__restrict__ in1,                        \
                                                       const type *__restrict__ in2,                        \
                                                       type *__restrict__ out, int n) {                     \
        using vtype = Vec<type, vlen>;                                                                      \
        constexpr const size_t alignment = sizeof(type)*vlen;                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                            \
        const int stride = blockDim.x * gridDim.x;                                                          \
        size_t in_pad = ALIGN_PAD_AMOUNT(in1, alignment);                                                   \
        const vtype * in1v = ALIGN_PTR(in1, alignment, const vtype*);                                       \
        const vtype * in2v = ALIGN_PTR(in2, alignment, const vtype*);                                       \
              vtype * outv = ALIGN_PTR(out, alignment, vtype*);                                             \
        for (int i = index; i < (n/vlen - in_pad/sizeof(type)); i += stride) {                              \
            outv[i] = vfn(in1v[i], in2v[i]);                                                                \
        }                                                                                                   \
        if (in_pad > 0) {                                                                                   \
            /* manage front values */                                                                       \
            if (index < ((in_pad/sizeof(type)) - 1)) {                                                      \
                out[index] = fn(in1[index], in2[index]);                                                    \
            }                                                                                               \
        }                                                                                                   \
        int remainder = (n%vlen);                                                                           \
        if (remainder > 0) {                                                                                \
            /* manage back values */                                                                        \
            if (index < (remainder-1)) {                                                                    \
                size_t idx = n - remainder + index;                                                         \
                out[idx] = fn(in1[idx], in2[idx]);                                                          \
            }                                                                                               \
        }                                                                                                   \
    }                                                                                                       \
    static __global__ void                                                                                  \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const type *__restrict__ in1,                          \
                                                     const type *__restrict__ in2,                          \
                                                     type *__restrict__ out, int n) {                       \
        /* non-vectorized version (e.g., due to mismatching alignment) */                                   \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                            \
        const int stride = blockDim.x * gridDim.x;                                                          \
        for (int i = index; i < n; i += stride) {                                                           \
            out[i] = fn(in1[i], in2[i]);                                                                    \
        }                                                                                                   \
    }                                                                                                       \
    void                                                                                                    \
    ompi_op_cuda_3buff_##name##_##type_name##_submit(const type *in1,                                       \
                                                     const type *in2,                                       \
                                                     type *out,                                             \
                                                     int count,                                             \
                                                     int threads_per_block,                                 \
                                                     int max_blocks,                                        \
                                                     hipStream_t stream) {                                     \
        int vcount  = (count + vlen-1)/vlen;                                                                \
        int threads = min(threads_per_block, vcount);                                                       \
        int blocks  = min((vcount + threads-1) / threads, max_blocks);                                      \
        int n = count;                                                                                      \
        hipStream_t s = stream;                                                                                \
        constexpr const size_t alignment = sizeof(type)*vlen;                                               \
        size_t in1_pad = ALIGN_PAD_AMOUNT(in1, alignment);                                                  \
        size_t in2_pad = ALIGN_PAD_AMOUNT(in2, alignment);                                                  \
        size_t out_pad = ALIGN_PAD_AMOUNT(out, alignment);                                                  \
        if (in1_pad == in2_pad && in1_pad == out_pad) {                                                     \
            ompi_op_cuda_3buff_##name##_##type_name##_kernel_v<<<blocks, threads, 0, s>>>(in1, in2, out, n);\
        } else {                                                                                            \
            ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads, 0, s>>>(in1, in2, out, n);  \
        }                                                                                                   \
    }
#else
#define VFUNC_FUNC(name, type_name, type, vlen, vfn, fn) FUNC_FUNC_FN(name, type_name, type, fn)
#endif // defined(USE_VECTORS)
/*
 * Since all the functions in this file are essentially identical, we
 * use a macro to substitute in names and types.  The core operation
 * in all functions that use this macro is the same.
 *
 * This macro is for minloc and maxloc
 */
#define LOC_FUNC_3BUF(name, type_name, op)                                                          \
    static __global__ void                                                                          \
    ompi_op_cuda_3buff_##name##_##type_name##_kernel(const ompi_op_predefined_##type_name##_t *__restrict__ in1, \
                                                     const ompi_op_predefined_##type_name##_t *__restrict__ in2, \
                                                     ompi_op_predefined_##type_name##_t *__restrict__ out,       \
                                                     int n)                                         \
    {                                                                                               \
        const int index = blockIdx.x * blockDim.x + threadIdx.x;                                    \
        const int stride = blockDim.x * gridDim.x;                                                  \
        for (int i = index; i < n; i += stride) {                                                   \
            const ompi_op_predefined_##type_name##_t *a1 = &in1[i];                                 \
            const ompi_op_predefined_##type_name##_t *a2 = &in2[i];                                 \
            ompi_op_predefined_##type_name##_t *b = &out[i];                                        \
            if (a1->v op a2->v) {                                                                   \
                b->v = a1->v;                                                                       \
                b->k = a1->k;                                                                       \
            } else if (a1->v == a2->v) {                                                            \
                b->v = a1->v;                                                                       \
                b->k = (a2->k < a1->k ? a2->k : a1->k);                                             \
            } else {                                                                                \
                b->v = a2->v;                                                                       \
                b->k = a2->k;                                                                       \
            }                                                                                       \
        }                                                                                           \
    }                                                                                               \
    void                                                                                            \
    ompi_op_cuda_3buff_##name##_##type_name##_submit(const ompi_op_predefined_##type_name##_t *in1, \
                                                     const ompi_op_predefined_##type_name##_t *in2, \
                                                     ompi_op_predefined_##type_name##_t *out,       \
                                                     int count,                                     \
                                                     int threads_per_block,                         \
                                                     int max_blocks,                                \
                                                     hipStream_t stream)                               \
    {                                                                                               \
        int threads = min(count, threads_per_block);                                                \
        int blocks  = min((count + threads-1) / threads, max_blocks);                               \
        ompi_op_cuda_3buff_##name##_##type_name##_kernel<<<blocks, threads,                         \
                                                           0, stream>>>(in1, in2, out, count);      \
    }


/*************************************************************************
 * Max
 *************************************************************************/

/* fixed-size types: 16B vector sizes
 * TODO: should this be fine-tuned to the architecture? */
 VFUNC_FUNC_3BUF(max,   int8_t,   int8_t, 16, vmax, tmax)
 VFUNC_FUNC_3BUF(max,  uint8_t,  uint8_t, 16, vmax, tmax)
 VFUNC_FUNC_3BUF(max,  int16_t,  int16_t, 8, vmax, tmax)
 VFUNC_FUNC_3BUF(max, uint16_t, uint16_t, 8, vmax, tmax)
 VFUNC_FUNC_3BUF(max,  int32_t,  int32_t, 4, vmax, tmax)
 VFUNC_FUNC_3BUF(max, uint32_t, uint32_t, 4, vmax, tmax)
 VFUNC_FUNC_3BUF(max,  int64_t,  int64_t, 2, vmax, tmax)
 VFUNC_FUNC_3BUF(max, uint64_t, uint64_t, 2, vmax, tmax)

 /* float */
 VFUNC_FUNC_3BUF(max, float, float, 4, vmax, tmax)
 VFUNC_FUNC_3BUF(max, double, double, 2, vmax, tmax)
 VFUNC_FUNC_3BUF(max, long_double, long double, 1, vmax, tmax)


/*************************************************************************
 * Min
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(min,   int8_t,   int8_t, 16, vmin, tmin)
VFUNC_FUNC_3BUF(min,  uint8_t,  uint8_t, 16, vmin, tmin)
VFUNC_FUNC_3BUF(min,  int16_t,  int16_t, 8, vmin, tmin)
VFUNC_FUNC_3BUF(min, uint16_t, uint16_t, 8, vmin, tmin)
VFUNC_FUNC_3BUF(min,  int32_t,  int32_t, 4, vmin, tmin)
VFUNC_FUNC_3BUF(min, uint32_t, uint32_t, 4, vmin, tmin)
VFUNC_FUNC_3BUF(min,  int64_t,  int64_t, 2, vmin, tmin)
VFUNC_FUNC_3BUF(min, uint64_t, uint64_t, 2, vmin, tmin)
VFUNC_FUNC_3BUF(min,  long,  long, 2, vmin, tmin)
VFUNC_FUNC_3BUF(min, ulong, unsigned long, 2, vmin, tmin)

/* float */
VFUNC_FUNC_3BUF(min, float, float, 4, vmin, tmin)
VFUNC_FUNC_3BUF(min, double, double, 2, vmin, tmin)
VFUNC_FUNC_3BUF(min, long_double, long double, 1, vmin, tmin)

/*************************************************************************
 * Sum
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(sum,   int8_t,   int8_t, 16, vsum, tsum)
VFUNC_FUNC_3BUF(sum,  uint8_t,  uint8_t, 16, vsum, tsum)
VFUNC_FUNC_3BUF(sum,  int16_t,  int16_t, 8, vsum, tsum)
VFUNC_FUNC_3BUF(sum, uint16_t, uint16_t, 8, vsum, tsum)
VFUNC_FUNC_3BUF(sum,  int32_t,  int32_t, 4, vsum, tsum)
VFUNC_FUNC_3BUF(sum, uint32_t, uint32_t, 4, vsum, tsum)
VFUNC_FUNC_3BUF(sum,  int64_t,  int64_t, 2, vsum, tsum)
VFUNC_FUNC_3BUF(sum, uint64_t, uint64_t, 2, vsum, tsum)
VFUNC_FUNC_3BUF(sum,  long,  long, 2, vsum, tsum)
VFUNC_FUNC_3BUF(sum, ulong, unsigned long, 2, vsum, tsum)

/* float */
VFUNC_FUNC_3BUF(sum, float, float, 4, vsum, tsum)
VFUNC_FUNC_3BUF(sum, double, double, 2, vsum, tsum)
VFUNC_FUNC_3BUF(sum, long_double, long double, 1, vsum, tsum)

/* Complex */
#undef current_func
#define current_func(a, b) (hipCaddf(a,b))
FUNC_FUNC_3BUF(sum, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCadd(a,b))
FUNC_FUNC_3BUF(sum, c_double_complex, hipDoubleComplex)

/*************************************************************************
 * Product
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(prod,   int8_t,   int8_t, 16, vprod, tprod)
VFUNC_FUNC_3BUF(prod,  uint8_t,  uint8_t, 16, vprod, tprod)
VFUNC_FUNC_3BUF(prod,  int16_t,  int16_t, 8, vprod, tprod)
VFUNC_FUNC_3BUF(prod, uint16_t, uint16_t, 8, vprod, tprod)
VFUNC_FUNC_3BUF(prod,  int32_t,  int32_t, 4, vprod, tprod)
VFUNC_FUNC_3BUF(prod, uint32_t, uint32_t, 4, vprod, tprod)
VFUNC_FUNC_3BUF(prod,  int64_t,  int64_t, 2, vprod, tprod)
VFUNC_FUNC_3BUF(prod, uint64_t, uint64_t, 2, vprod, tprod)
VFUNC_FUNC_3BUF(prod,  long,  long, 2, vprod, tprod)
VFUNC_FUNC_3BUF(prod, ulong, unsigned long, 2, vprod, tprod)

/* float */
VFUNC_FUNC_3BUF(prod, float, float, 4, vprod, tprod)
VFUNC_FUNC_3BUF(prod, double, double, 2, vprod, tprod)
VFUNC_FUNC_3BUF(prod, long_double, long double, 1, vprod, tprod)

/* Complex */
#undef current_func
#define current_func(a, b) (hipCmulf(a,b))
FUNC_FUNC_3BUF(prod, c_float_complex, hipFloatComplex)
#undef current_func
#define current_func(a, b) (hipCmul(a,b))
FUNC_FUNC_3BUF(prod, c_double_complex, hipDoubleComplex)

/*************************************************************************
 * Logical AND
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(land,   int8_t,   int8_t, 16, vland, tland)
VFUNC_FUNC_3BUF(land,  uint8_t,  uint8_t, 16, vland, tland)
VFUNC_FUNC_3BUF(land,  int16_t,  int16_t, 8, vland, tland)
VFUNC_FUNC_3BUF(land, uint16_t, uint16_t, 8, vland, tland)
VFUNC_FUNC_3BUF(land,  int32_t,  int32_t, 4, vland, tland)
VFUNC_FUNC_3BUF(land, uint32_t, uint32_t, 4, vland, tland)
VFUNC_FUNC_3BUF(land,  int64_t,  int64_t, 2, vland, tland)
VFUNC_FUNC_3BUF(land, uint64_t, uint64_t, 2, vland, tland)
VFUNC_FUNC_3BUF(land,  long,  long, 2, vland, tland)
VFUNC_FUNC_3BUF(land, ulong, unsigned long, 2, vland, tland)

/* C++ bool */
VFUNC_FUNC_3BUF(land, bool, bool, 16, vland, tland)

/*************************************************************************
 * Logical OR
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(lor,   int8_t,   int8_t, 16, vlor, tlor)
VFUNC_FUNC_3BUF(lor,  uint8_t,  uint8_t, 16, vlor, tlor)
VFUNC_FUNC_3BUF(lor,  int16_t,  int16_t, 8, vlor, tlor)
VFUNC_FUNC_3BUF(lor, uint16_t, uint16_t, 8, vlor, tlor)
VFUNC_FUNC_3BUF(lor,  int32_t,  int32_t, 4, vlor, tlor)
VFUNC_FUNC_3BUF(lor, uint32_t, uint32_t, 4, vlor, tlor)
VFUNC_FUNC_3BUF(lor,  int64_t,  int64_t, 2, vlor, tlor)
VFUNC_FUNC_3BUF(lor, uint64_t, uint64_t, 2, vlor, tlor)
VFUNC_FUNC_3BUF(lor,  long,  long, 2, vlor, tlor)
VFUNC_FUNC_3BUF(lor, ulong, unsigned long, 2, vlor, tlor)

/* C++ bool */
VFUNC_FUNC_3BUF(lor, bool, bool, 16, vlor, tlor)

/*************************************************************************
 * Logical XOR
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(lxor,   int8_t,   int8_t, 16, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor,  uint8_t,  uint8_t, 16, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor,  int16_t,  int16_t, 8, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor, uint16_t, uint16_t, 8, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor,  int32_t,  int32_t, 4, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor, uint32_t, uint32_t, 4, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor,  int64_t,  int64_t, 2, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor, uint64_t, uint64_t, 2, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor,  long,  long, 2, vlxor, tlxor)
VFUNC_FUNC_3BUF(lxor, ulong, unsigned long, 2, vlxor, tlxor)

/* C++ bool */
VFUNC_FUNC_3BUF(lxor, bool, bool, 16, vlxor, tlxor)


/*************************************************************************
 * Bitwise AND
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(band,   int8_t,   int8_t, 16, vband, tband)
VFUNC_FUNC_3BUF(band,  uint8_t,  uint8_t, 16, vband, tband)
VFUNC_FUNC_3BUF(band,  int16_t,  int16_t, 8, vband, tband)
VFUNC_FUNC_3BUF(band, uint16_t, uint16_t, 8, vband, tband)
VFUNC_FUNC_3BUF(band,  int32_t,  int32_t, 4, vband, tband)
VFUNC_FUNC_3BUF(band, uint32_t, uint32_t, 4, vband, tband)
VFUNC_FUNC_3BUF(band,  int64_t,  int64_t, 2, vband, tband)
VFUNC_FUNC_3BUF(band, uint64_t, uint64_t, 2, vband, tband)
VFUNC_FUNC_3BUF(band,  long,  long, 2, vband, tband)
VFUNC_FUNC_3BUF(band, ulong, unsigned long, 2, vband, tband)

/* C++ byte */
VFUNC_FUNC_3BUF(band, byte, char, 16, vband, tband)

/*************************************************************************
 * Bitwise OR
 *************************************************************************/

/* C integer */
VFUNC_FUNC_3BUF(bor,   int8_t,   int8_t, 16, vbor, tbor)
VFUNC_FUNC_3BUF(bor,  uint8_t,  uint8_t, 16, vbor, tbor)
VFUNC_FUNC_3BUF(bor,  int16_t,  int16_t, 8, vbor, tbor)
VFUNC_FUNC_3BUF(bor, uint16_t, uint16_t, 8, vbor, tbor)
VFUNC_FUNC_3BUF(bor,  int32_t,  int32_t, 4, vbor, tbor)
VFUNC_FUNC_3BUF(bor, uint32_t, uint32_t, 4, vbor, tbor)
VFUNC_FUNC_3BUF(bor,  int64_t,  int64_t, 2, vbor, tbor)
VFUNC_FUNC_3BUF(bor, uint64_t, uint64_t, 2, vbor, tbor)
VFUNC_FUNC_3BUF(bor,  long,  long, 2, vbor, tbor)
VFUNC_FUNC_3BUF(bor, ulong, unsigned long, 2, vbor, tbor)

/* C++ byte */
VFUNC_FUNC_3BUF(bor, byte, char, 16, vbor, tbor)

/*************************************************************************
 * Max location
 *************************************************************************/

LOC_FUNC_3BUF(maxloc, float_int, >)
LOC_FUNC_3BUF(maxloc, double_int, >)
LOC_FUNC_3BUF(maxloc, long_int, >)
LOC_FUNC_3BUF(maxloc, 2int, >)
LOC_FUNC_3BUF(maxloc, short_int, >)
LOC_FUNC_3BUF(maxloc, long_double_int, >)

/* Fortran compat types */
LOC_FUNC_3BUF(maxloc, 2float, >)
LOC_FUNC_3BUF(maxloc, 2double, >)
LOC_FUNC_3BUF(maxloc, 2int8, >)
LOC_FUNC_3BUF(maxloc, 2int16, >)
LOC_FUNC_3BUF(maxloc, 2int32, >)
LOC_FUNC_3BUF(maxloc, 2int64, >)

/*************************************************************************
 * Min location
 *************************************************************************/

LOC_FUNC_3BUF(minloc, float_int, <)
LOC_FUNC_3BUF(minloc, double_int, <)
LOC_FUNC_3BUF(minloc, long_int, <)
LOC_FUNC_3BUF(minloc, 2int, <)
LOC_FUNC_3BUF(minloc, short_int, <)
LOC_FUNC_3BUF(minloc, long_double_int, <)

/* Fortran compat types */
LOC_FUNC_3BUF(minloc, 2float, <)
LOC_FUNC_3BUF(minloc, 2double, <)
LOC_FUNC_3BUF(minloc, 2int8, <)
LOC_FUNC_3BUF(minloc, 2int16, <)
LOC_FUNC_3BUF(minloc, 2int32, <)
LOC_FUNC_3BUF(minloc, 2int64, <)
