#include "hip/hip_runtime.h"
/*
 *  Copyright (C) by Argonne National Laboratory.
 *      See COPYRIGHT in top-level directory.
 */

#include "mpl_gpu_cuda.h"
#include <stdio.h>

__global__ void MPL_gpu_kernel_trigger(MPL_gpu_event_t *var)
{
    *var -= 1;
    __threadfence_system();
}

__global__ void MPL_gpu_kernel_wait(MPL_gpu_event_t *var)
{
    while(*var > 0);
}

extern "C"
void MPL_gpu_enqueue_trigger(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;
    void *args[] = {&var};
    cerr = hipLaunchKernel((const void *) MPL_gpu_kernel_trigger, dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}

extern "C"
void MPL_gpu_enqueue_wait(volatile int *var, hipStream_t stream)
{
    hipError_t cerr;

    void *args[] = {&var};
    cerr = hipLaunchKernel((const void *) MPL_gpu_kernel_wait, dim3(1,1,1), dim3(1,1,1),
                            args, 0, stream);
    if (cerr != hipSuccess) {
        fprintf(stderr, "CUDA Error (%s): %s\n", __func__, hipGetErrorString(cerr));
    }
}

extern "C"
void MPL_gpu_event_init_count(MPL_gpu_event_t *var, int count)
{
    *var = count;
}

extern "C"
void MPL_gpu_event_complete(MPL_gpu_event_t *var)
{
    *var -= 1;
}

extern "C"
bool MPL_gpu_event_is_complete(MPL_gpu_event_t *var)
{
    return (*var) <= 0;
}
